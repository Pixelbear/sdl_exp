#include "hip/hip_runtime.h"
#include "EntityScene.h"

__global__ void moveKernel(float *data, float i, hipTextureObject_t texObj)
{
    unsigned int id = threadIdx.x;
    float id2 = (id*3.6)+i;
    id2 = (id2 >= 360) ? id2-360 : id2;
    data[(id * 4) + 0] = 100 * (float)sinf(id2*0.0174533);
    data[(id * 4) + 2] = 100 * (float)cosf(id2*0.0174533);
}
void EntityScene::hipInit()
{
    float *tempData = (float*)malloc(sizeof(float) * 4 * 100);
    for (int i = 0; i < 100; i++)
    {
        tempData[(i * 4) + 0] = 100 * sin(i*3.6);
        tempData[(i * 4) + 1] = -50.0f;
        tempData[(i * 4) + 2] = 100 * cos(i*3.6);
    }
    hipMemcpy(this->cuTexBuf->d_mappedPointer, tempData, sizeof(float) * 4 * 100, hipMemcpyHostToDevice);
    hipError_t status = hipGetLastError();
    if (status != hipSuccess || (status = hipGetLastError()) != hipSuccess)
    {
        if (status == hipErrorUnknown)
        {
            printf("An Unknown CUDA Error Occurred :(\n");
            printf("Perhaps performing the same operation under the CUDA debugger with Memory Checker enabled could help!\n");
            printf("If this error only occurs outside of NSight debugging sessions, or causes the system to lock up. It may be caused by not passing the required amount of shared memory to a kernal launch that uses runtime sized shared memory.\n");
            printf("Also possible you have forgotten to allocate texture memory you are trying to read\n");
            printf("Passing a buffer to 'hipGraphicsSubResourceGetMappedArray' or a texture to 'hipGraphicsResourceGetMappedPointer'.\n");
            getchar();
            exit(1);
        }
        printf(" CUDA Error Occurred;\n%s\n", hipGetErrorString(status));
        getchar();
        exit(1);
    }
    free(tempData);
}

void EntityScene::cuUpdate()
{
    static float i = 0;
    i-=0.05f;
    i = (i < 0) ? 359 : i;
    moveKernel << <1, 100 >> >(this->cuTexBuf->d_mappedPointer, i, this->cuTexBuf->cuTextureObj); 
    hipDeviceSynchronize(); 
    hipError_t status = hipGetLastError();
    if (status != hipSuccess || (status = hipGetLastError()) != hipSuccess)
    {
        if (status == hipErrorUnknown)
        {
            printf("An Unknown CUDA Error Occurred :(\n");
            printf("Perhaps performing the same operation under the CUDA debugger with Memory Checker enabled could help!\n");
            printf("If this error only occurs outside of NSight debugging sessions, or causes the system to lock up. It may be caused by not passing the required amount of shared memory to a kernal launch that uses runtime sized shared memory.\n");
            printf("Also possible you have forgotten to allocate texture memory you are trying to read\n");
            printf("Passing a buffer to 'hipGraphicsSubResourceGetMappedArray' or a texture to 'hipGraphicsResourceGetMappedPointer'.\n");
            getchar();
            exit(1);
        }
        printf(" CUDA Error Occurred;\n%s\n",  hipGetErrorString(status));
        getchar();
        exit(1);
    }
}
